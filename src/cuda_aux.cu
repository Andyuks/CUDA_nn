#include <hip/hip_runtime.h>


void cuda_copyToDev(void *dst, const void *src, size_t count){
    hipMemcpy(dst, src, count, hipMemcpyHostToDevice);
}

void cuda_copyToHost(void* dst, const void* src, size_t count){
    hipMemcpy(dst, src, count, hipMemcpyDeviceToHost);
}

void cuda_setDevice(int id){
	hipSetDevice(id);
}

void cuda_getDeviceCount(int* gpu_n){
	hipGetDeviceCount(gpu_n);
}