#include "hip/hip_runtime.h"
/* matrix.cu */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "nn_aux.h"

#ifdef TIMING
    #include <time.h>
    #include "utils.h"
#endif

#include "matrix.cuh"
#include "globals.cuh"




/* GPU: alloc matrix 2V*/
double **cuda_alloc_matrix_2v(int n_layers, int *size, int *size_prev, double (*init_weight_ptr)(void)){
    double **m;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    hipError_t malloc_call;
    malloc_call = hipMalloc(&m, n_layers * sizeof(double*));
    
    if (malloc_call != hipSuccess)
        return NULL;

    if(idx < n_layers) {
        malloc_call = hipMalloc(&m[idx], size[idx] * size_prev[idx] * sizeof(double));
        if (malloc_call != hipSuccess) {
            cuda_matrix_free_2D(m, n_layers);
            return NULL;
        }
    }
    
    if(idx < (n_layers * size[idx] * size_prev[idx])) 
		(*m)[idx] = init_weight_ptr();
       
    return m;
    
}


/* GPU: alloc matrix 1V*/
double **cuda_alloc_matrix_1v(int n_layers, int *size, double (*init_weight_ptr)(void)) {
    double **m;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    hipError_t malloc_call;
    malloc_call = hipMalloc(&m, n_layers * sizeof(double*));
    
    if (malloc_call != hipSuccess)
        return NULL;

    if(idx < n_layers) {
        malloc_call = hipMalloc(&m[idx], size[idx] * sizeof(double));
        if (malloc_call != hipSuccess) {
            cuda_matrix_free_2D(m, n_layers);
            return NULL;
        }
    }
    
    if(idx < (n_layers * size[idx])) 
		(*m)[idx] = init_weight_ptr();
       
    return m;
}


/* GPU: alloc array */
double *cuda_alloc_array(int length) {

    double *v;
    int idx = threadIdx.x + blockIdx.x * blockDim.x; 

    hipError_t malloc_call;
    malloc_call = hipMalloc(&v, length * sizeof(double));
    
    if (malloc_call != hipSuccess)
        return NULL;

    if(idx < length)
        v[idx] = 0.0;
    
    return(v);
}




/* GPU: alloc matrix */
double *cuda_alloc_matrix(int rows, int cols) {

    double *m;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    hipError_t malloc_call;
    malloc_call = hipMalloc(&m, rows * cols * sizeof(double));
    
    if (malloc_call != hipSuccess)
        return NULL;

    if(idx < (rows * cols))
        m[idx] = 0.0;
    
    return(m);
}



/* GPU: matrix free 2D */
void matrix_free_2D(double **m, int n_layers) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if(idx < n_layers)
        if (m[idx] != NULL)
            hipFree(m[idx]);

    hipFree(m);
}


/* GPU: matrix free */
void cuda_matrix_free(double *m){

    if (m != NULL)
        hipFree(m);
}



double *m_elem(double *m, int length, int x, int y){

    return (double*)&m[length * x + y];
}



/* operations */ 

/* GPU: addition of matrix */
__global__ void cuda_matrix_sum(double *C, double *A, double *B, int rows, int cols) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x; 
	if(idx < (rows * cols)) /*ensure threads not outside dim*/
		C[idx] = A[idx] + B[idx];
}



/* GPU: substraction of matrix  */
__global__ void cuda_matrix_sub(double *C, double *A, double *B, int rows, int cols) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x; 
	if(idx < (rows * cols)) /*ensure threads not outside dim*/
		C[idx] = A[idx] - B[idx];
}



/* GPU:  mul cnt  */
__global__ void cuda_matrix_mul_cnt(double *m, int rows, int cols, double cnt) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x; 
	if(idx < (rows * cols)) /*ensure threads not outside dim*/
		m[idx] *= cnt;
}



/* GPU:  zero  */
__global__ void cuda_matrix_zero(double *m, int rows, int cols) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x; 
	if(idx < (rows * cols)) /*ensure threads not outside dim*/
		m[idx] = 0.0;
}



/* GPU: cuda matrix mul dot  */
__global__ void cuda_matrix_mul_dot(double *C, double *A, double *B, int rows, int cols) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x; 
	if(idx < (rows * cols)) /*ensure threads not outside dim*/
		C[idx] = A[idx] * B[idx];
}

/* GPU: matrix transpose (OPERATIONS)*/
__global__ double * cuda_matrix_transpose_op(double * m, double * m_tr, int rows, int cols) {

	idx = threadIdx.x + blockIdx.x * blockDim.x;
	i = idx / cols;
	j = idx % cols;

	m_tr[idx] = m[j * blockDim.x + i];
    return(m_tr);
}

/* GPU: matrix transpose */
double * cuda_matrix_transpose(double * m, int rows, int cols, size_t N) {
    double *m_tr;
    
    // Allocate memory in Host (CPU)
    hipError_t malloc_call;
    malloc_call = hipHostMalloc(&m_tr, rows * cols * sizeof(double));
    
    if (malloc_call != hipSuccess)
        return NULL;

    // Division of function due to illegal malloc call inside global function
    int thr_per_blk = THR_PER_BLOCK;
    int blk_in_grid = ceil( (float)N / thr_per_blk );

	cuda_matrix_transpose_op <<<blk_in_grid, thr_per_blk>>>(m, m_tr, rows, cols);
    return(m_tr);
}

/* GPU: cuda matrix mul */
__global__ void cuda_matrix_mul(double *C, double *A, double *B, int a_rows, int a_cols, int b_rows, int b_cols) {
	assert(a_cols == b_rows);
    double sum = 0.0;
    int i;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ double c_aux[THR_PER_BLOCK];
    
#ifdef TIMING
    int res_time;
    struct timespec t1, t2;
    clockid_t clk_id = CLOCK_MONOTONIC;
    res_time = clock_gettime(clk_id, &t1);
#endif


    if (idx < a_rows * b_cols) {
        c_aux[threadIdx.x] = A[idx] * B[idx]; /* need index inside block */
        __syncthreads();

        if(threadIdx.x == 0) {
            for(i = 0; i < THR_PER_BLOCK; i++) // TODO: mirar si optimizar op cambiando THR_PER_BLOCK
                sum += c_aux[i];
            atomicAdd(C, sum);
        }
    }
	
#ifdef TIMING
    res_time = clock_gettime(clk_id, &t2);
    printf("Matrix mul execution time: %ld us \n", diff_time(t2, t1));
#endif
}



/* matrix multiplication add */

__global__ void cuda_matrix_mul_add(double *C, double *A, double *B, int a_rows, int a_cols, int b_rows, int b_cols, double *D) {
	assert(a_cols == b_rows);
    double sum = 0.0;
    int i;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ double c_aux[THR_PER_BLOCK];
    
#ifdef TIMING
    int res_time;
    struct timespec t1, t2;
    clockid_t clk_id = CLOCK_MONOTONIC;
    res_time = clock_gettime(clk_id, &t1);
#endif


    if (idx < a_rows * b_cols) {
        c_aux[threadIdx.x] = A[idx] * B[idx]; /* need index inside block */
        __syncthreads();

        if(threadIdx.x == 0) {
            for(i = 0; i < THR_PER_BLOCK; i++) // TODO: mirar si optimizar op cambiando THR_PER_BLOCK
                sum += c_aux[i];
            sum += D[idx];
            atomicAdd(C, sum);
        }
    }
	
#ifdef TIMING
    res_time = clock_gettime(clk_id, &t2);
    printf("Matrix mul execution time: %ld us \n", diff_time(t2, t1));
#endif
}




/* GPU:  apply fun to matrix */
__global__ void cuda_matrix_func (double *A, double *B, int rows, int cols, double (*func)(double)) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x; 
	if(idx < (rows * cols)) /*ensure threads not outside dim*/
		A[idx] = func(B[idx]);
}


/* print matrix */
void print_matrix(double *m, int m_rows, int m_cols)
{
    int col, row;
    printf("%d %d\n", m_rows, m_cols);
    for (row = 0; row < m_rows; row++){
        for(col = 0; col < m_cols; col++){
            printf("(%d %d) %.*lf ", row, col, 10, *m_elem(m, m_cols, row, col));
        }
        printf("\n");
    }
    printf("\n");
}

